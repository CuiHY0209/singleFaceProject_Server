
#include <hip/hip_runtime.h>
﻿//#include <iostream>
//#include <cstdio>
//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <opencv2/core.hpp>
//#include <opencv2/core/cuda.hpp>
//#include <opencv2/opencv.hpp>
//#include <opencv2/highgui.hpp>
//#include "RGBD2EIA.h"
//
//#define NUM_PATTERNS 8
//#define NUM_PI 3.14159
//
//long timeFlag[10] = { 0 };
//
/////***
//// *** 5张视差图并行计算生成视差图
////*/
////__global__ void CalcParaImageKernel(cv::cuda::PtrStepSz<uchar> depthMap_gpu, cv::cuda::PtrStepSz<uchar3> RGBMap_gpu, double paraAngle,
////	int cameraRows, int cameraCols, int depthThreshold, cv::cuda::PtrStepSz<uchar3>* ParaMat_gpu)
////{
////
////	//int coorX = blockIdx.x * blockDim.x + threadIdx.x;
////	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
////	int iCamera = blockIdx.z;
////
////	printf("thread %d successfully ran\n", coorY);
////
////	/*不满足阈值赋值为0*/
////	uchar3 zeros;
////	zeros.x = 0; zeros.y = 0; zeros.z = 0;
////	if (iCamera > blockDim.z / 2) {
////		/*按列遍历，每列同时异步处理*/
////		for (int iCoorX = 0; iCoorX < depthMap_gpu.cols; iCoorX++)
////		{
////			int coorParaX = (iCoorX - (int(depthMap_gpu.ptr(coorY)[iCoorX]) - 50) * paraAngle * (iCamera - cameraCols / 2));
////			int coorParaY = coorY;
////
////			if (iCoorX >= depthMap_gpu.cols || iCoorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
////			if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
////
////
////
////			/*如果小于深度阈值，则该像素直接舍弃*/
////			if (depthMap_gpu.ptr(coorY)[iCoorX] < depthThreshold) {
////				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = zeros;
////			}
////			else {
////				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[iCoorX];
////
////				/*空洞填补*/
////				if ((ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].x == zeros.x) &&
////					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].y == zeros.y) &&
////					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].z == zeros.z)) {
////					if (coorParaX >= 1) {
////						ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX - 1];
////					}
////				}
////			}
////		}
////	}
////	else {
////		/*按列遍历，每列同时异步处理*/
////		for (int iCoorX = depthMap_gpu.cols - 1; iCoorX >= 0; iCoorX--)
////		{
////			int coorParaX = (iCoorX - (int(depthMap_gpu.ptr(coorY)[iCoorX]) - 50) * paraAngle * (iCamera - cameraCols / 2));
////			int coorParaY = coorY;
////
////			if (iCoorX >= depthMap_gpu.cols || iCoorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
////			if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
////
////
////
////			/*如果小于深度阈值，则该像素直接舍弃*/
////			if (depthMap_gpu.ptr(coorY)[iCoorX] < depthThreshold) {
////				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = zeros;
////			}
////			else {
////				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[iCoorX];
////
////				/*空洞填补*/
////				if ((ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].x == zeros.x) &&
////					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].y == zeros.y) &&
////					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].z == zeros.z)) {
////					if (coorParaX >= 1) {
////						ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX - 1];
////					}
////				}
////			}
////		}
////	}
////
////
////}
//
///***
// *** 正向遍历生成视差图
//*/
//__global__ void CalcParaImageKernel1(cv::cuda::PtrStepSz<uchar> depthMap_gpu, cv::cuda::PtrStepSz<uchar3> RGBMap_gpu, double paraAngle,
//	int cameraRows, int cameraCols, int iCamera, int depthThreshold, cv::cuda::PtrStepSz<uchar3> ParaMat_gpu)
//{
//	
//	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
//	///*测试用*/
//	//printf("thread %d successfully ran\n", coorY);
//
//	/*不满足阈值赋值为0*/
//	uchar3 zeros;
//	zeros.x = 0; zeros.y = 0; zeros.z = 0;
//	/*按列遍历，每列同时异步处理*/
//	int coorParaX = (coorX - (int(depthMap_gpu.ptr(coorY)[coorX]) - 50) * paraAngle * (iCamera - cameraCols / 2));
//	int coorParaY = coorY;
//
//	if (coorX >= depthMap_gpu.cols || coorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
//	if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
//		
//
//	/*如果小于深度阈值，则该像素直接舍弃*/
//	if (depthMap_gpu.ptr(coorY)[coorX] < depthThreshold) {
//		ParaMat_gpu.ptr(coorParaY)[coorParaX] = zeros;
//	}
//	else {
//		ParaMat_gpu.ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[coorX];
//	}		
//}
//
///***
// *** 反向遍历生成视差图
//*/
//__global__ void CalcParaImageKernel2(cv::cuda::PtrStepSz<uchar> depthMap_gpu, cv::cuda::PtrStepSz<uchar3> RGBMap_gpu, double paraAngle,
//	int cameraRows, int cameraCols, int iCamera, int depthThreshold, cv::cuda::PtrStepSz<uchar3> ParaMat_gpu)
//{
//	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
//	/*不满足阈值赋值为0*/
//	uchar3 zeros;
//	zeros.x = 0; zeros.y = 0; zeros.z = 0;
//
//	int coorParaX = (coorX - (int(depthMap_gpu.ptr(coorY)[coorX]) - 10) * paraAngle * (iCamera - cameraCols / 2));
//	int coorParaY = coorY;
//
//	if (coorX >= depthMap_gpu.cols || coorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
//	if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
//
//	/*如果小于深度阈值，则该像素直接舍弃*/
//	if (depthMap_gpu.ptr(coorY)[coorX] < depthThreshold) {
//		ParaMat_gpu.ptr(coorParaY)[coorParaX] = zeros;
//	}
//	else {
//		ParaMat_gpu.ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[coorX];
//	}	
//}
//
///*空洞填补*/
//__global__ void FillVacuum_backward(cv::cuda::PtrStepSz<uchar3> ParaMat_gpu) {
//	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
//
//	uchar3 zeros;
//	zeros.x = 0; zeros.y = 0; zeros.z = 0;
//
//	if ((ParaMat_gpu.ptr(coorY)[coorX].x == zeros.x)&&
//		(ParaMat_gpu.ptr(coorY)[coorX].y == zeros.y)&&
//		(ParaMat_gpu.ptr(coorY)[coorX].z == zeros.z)) {
//		if (coorX < ParaMat_gpu.cols - 1) {
//			ParaMat_gpu.ptr(coorY)[coorX] = ParaMat_gpu.ptr(coorY)[coorX + 1];
//		}		
//	}
//}
//__global__ void FillVacuum_forward(cv::cuda::PtrStepSz<uchar3> ParaMat_gpu) {
//	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
//
//	uchar3 zeros;
//	zeros.x = 0; zeros.y = 0; zeros.z = 0;
//
//	if ((ParaMat_gpu.ptr(coorY)[coorX].x == zeros.x) &&
//		(ParaMat_gpu.ptr(coorY)[coorX].y == zeros.y) &&
//		(ParaMat_gpu.ptr(coorY)[coorX].z == zeros.z)) {
//		if (coorX > 0) {
//			ParaMat_gpu.ptr(coorY)[coorX] = ParaMat_gpu.ptr(coorY)[coorX - 1];
//		}
//	}
//}
//
//
//
///*----------------------------
//* 功能 : 读入RGB处理成Stereo cv::Mat-------------------非整数视点代码，记得注释掉整数视点代码，就在上面
//*----------------------------
//* 函数 : RemapRGBMat2StereoMat_KERNEL
//* 访问 : public
//* 返回 : 0 - 操作失败，1 - 操作成功
//*
//* 参数 : inRGBMat_CUDA		[in]		输入视差图序列（RGB格式）
//* 参数 : iCurCam			[in]		当前相机编号
//* 参数 : effCamNum			[in]		有效相机总数
//* 参数 : slope				[in]		斜率
//* 参数 : outStereoMat_CUDA	[in/out]	输出，处理完图像(StereoMat)
//*/
//__global__ void RemapRGBMat2StereoMat_KERNEL(cv::cuda::PtrStepSz<uchar3> inRGBMat_CUDA, int iCurCam, double effCamNum, double slope, cv::cuda::PtrStepSz<uchar3> outStereoMat_CUDA)
//{
//	// 遍历子坐标
//	int coorDim_X = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorDim_Y = blockIdx.y * blockDim.y + threadIdx.y;
//
//	// 输入图像（视差图）坐标
//	int coorIn_X = coorDim_X / 3 * inRGBMat_CUDA.cols / outStereoMat_CUDA.cols;
//	int coorIn_Y = coorDim_Y * inRGBMat_CUDA.rows / outStereoMat_CUDA.rows;
//
//	// 输出图像（合成图像）坐标
//	int coorOut_X = coorDim_X / 3;
//	int coorOut_Y = coorDim_Y;
//	int isubx = coorDim_X % 3;
//
//	float coordinate = 0.000000f;
//
//	//非整数视点个数
//	float effViewpointNum = 4.666666f;
//
//
//	if (coorDim_Y % 2 == 0)
//	{
//		coordinate = coorDim_X + coorDim_Y * 3 * slope + outStereoMat_CUDA.cols * 3;
//	}
//	else
//	{
//		coordinate = coorDim_X + coorDim_Y * 3 * slope + outStereoMat_CUDA.cols * 3 + (float)1 / 3;
//	}
//
//
//
//	int ifactor = effCamNum - 1 - floor(fmodf(coordinate, effViewpointNum));
//
//
//	if (ifactor == iCurCam)
//	{
//		if (coorIn_X >= 0 && coorIn_X < inRGBMat_CUDA.cols && coorIn_Y >= 0 && coorIn_Y < inRGBMat_CUDA.rows)
//		{
//			if (coorOut_X >= 0 && coorOut_X < outStereoMat_CUDA.cols && coorOut_Y >= 0 && coorOut_Y < outStereoMat_CUDA.rows)
//			{
//				if (isubx == 0)
//				{
//					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].x = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].x;
//				}
//				else if (isubx == 1)
//				{
//					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].y = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].y;
//				}
//				else
//				{
//					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].z = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].z;
//				}
//			}
//		}
//	}
//}
//
//extern "C" bool genEIAImgs_CUDA(cv::cuda::GpuMat& RGBMap_gpu, cv::cuda::GpuMat& depthMap_gpu,HARDWAREPARA& hardwarePara, cv::cuda::GpuMat* &paraImages_gpu, cv::cuda::GpuMat& eiaImage_gpu)
//{	
//	timeFlag[2] = clock();
//	///*方法一，五张视差图并行处理*/
//	///*未调通*/
//	//dim3 blockSize(4, depthMap_gpu.rows / 5);
//	//dim3 gridSize(depthMap_gpu.cols / blockSize.x, 5, hardwarePara._VIEWNUM);
//
//	///*测试用*/
//	//cv::Mat Para_obs = cv::Mat::zeros(paraImages_gpu[0].size(), CV_8UC3);
//	//paraImages_gpu[0].download(Para_obs);
//	//cv::imwrite("output/observerPara.jpg", Para_obs);
//
//	///*合成视差图*/
//	//cv::cuda::PtrStepSz<uchar3>* paraImages_gpu_PtrStepSz = new cv::cuda::PtrStepSz<uchar3>[hardwarePara._VIEWNUM];
//	//for (int i = 0; i < hardwarePara._VIEWNUM; i++) {
//	//	paraImages_gpu_PtrStepSz[i] = paraImages_gpu[i];
//	//}
//	//CalcParaImageKernel << <gridSize, blockSize >> > (depthMap_gpu, RGBMap_gpu, hardwarePara._DISPARITYANGLE, 1, hardwarePara._VIEWNUM,
//	//	hardwarePara._DEPTHTHRESHOLD, paraImages_gpu_PtrStepSz);
//	//cudaDeviceSynchronize();
//
//	///*测试用*/				
//	//paraImages_gpu[0].download(Para_obs);
//	//cv::imwrite("output/observerPara.jpg", Para_obs);
//	//for (int iCamera = 0; iCamera < hardwarePara._VIEWNUM; iCamera++)
//	//{
//	//	/*将视差图RGB映射到EIA*/
//	//	RemapRGBMat2StereoMat_CUDA(paraImages_gpu[iCamera], iCamera, hardwarePara, eiaImage_gpu);
//	//	/*测试用*/
//	//	cv::Mat eia_obs = cv::Mat::zeros(eiaImage_gpu.size(), CV_8UC3);
//	//	eiaImage_gpu.download(eia_obs);
//	//	cv::imwrite("output/observerEIA.jpg", eia_obs);
//	//}
//
//	/*方法二，五张视差图串行处理*/
//	for (int iCamera = 0; iCamera < hardwarePara._VIEWNUM; iCamera++)
//	{
//		
//
//		/*合成视差图*/
//		timeFlag[4] = clock();
//		dim3 blockSize(4, depthMap_gpu.rows / 5);
//		dim3 gridSize(depthMap_gpu.cols / blockSize.x, 5);
//
//		// do it in Kernel
//		CalcParaImageKernel1 << <gridSize, blockSize >> > (depthMap_gpu, RGBMap_gpu, hardwarePara._DISPARITYANGLE, 1, hardwarePara._VIEWNUM,
//			iCamera, hardwarePara._DEPTHTHRESHOLD, paraImages_gpu[iCamera]);
//
//		timeFlag[5] = clock();
//		/*空洞填补*/
//		for (int iFillTimes = 0; iFillTimes < 4; iFillTimes++) {
//			for (int iParaImg = 0; iParaImg < hardwarePara._VIEWNUM; iParaImg++) {
//				if (iParaImg < hardwarePara._VIEWNUM / 2) {
//					FillVacuum_backward << <gridSize, blockSize >> > (paraImages_gpu[iParaImg]);
//				}
//				else {
//					FillVacuum_forward << <gridSize, blockSize >> > (paraImages_gpu[iParaImg]);
//				}
//			}
//		}		
//		timeFlag[6] = clock();
//		
//		/*将视差图RGB映射到EIA*/
//		RemapRGBMat2StereoMat_CUDA(paraImages_gpu[iCamera], iCamera, hardwarePara, eiaImage_gpu);
//		//cudaDeviceSynchronize();
//		
//		timeFlag[7] = clock();
//	}	
//	return true;
//}
//
///*----------------------------
// * 功能 : 读入RGB处理成II Eia cv::Mat
// *----------------------------
// * 函数 : RemapRGBMat2EiaMat_CUDA
// * 访问 : public
// * 返回 : 0 - 操作失败，1 - 操作成功
// *
// * 参数 : inRGBMat_GPU		[in]		输入视差图序列（RGB格式）
// * 参数 : iCurCam			[in]		当前相机编号
// * 参数 : camAayPer			[in]		相机及显示配置参数
// * 参数 : outEiaMat_GPU		[in/out]	输出，处理完图像(EiaMat)
// */
//extern "C" bool RemapRGBMat2StereoMat_CUDA(cv::cuda::GpuMat& inRGBMat_GPU, int iCurCam, HARDWAREPARA& hardwarePara, cv::cuda::GpuMat& outEiaMat_GPU)
//{
//	dim3 blockSize(30, 30);
//	int pixel_W = outEiaMat_GPU.cols * outEiaMat_GPU.channels();;
//	int pixel_H = outEiaMat_GPU.rows;
//	dim3 gridSize(pixel_W/ blockSize.x, pixel_H /blockSize.y);
//
//	RemapRGBMat2StereoMat_KERNEL << <gridSize, blockSize >> > (inRGBMat_GPU, iCurCam, hardwarePara._EFFICIENTVIEWNUM
//		, hardwarePara._EFFICIENTSLOPE, outEiaMat_GPU);
//	return true;
//}
//
//
