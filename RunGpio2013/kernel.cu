#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include "RGBD2EIA.h"

#define NUM_PATTERNS 8
#define NUM_PI 3.14159

long timeFlag[20] = { 0 };

///*并行生成多张视差图*/
//__global__ void CalcParaImageKernel(cv::cuda::PtrStepSz<uchar> depthMap_gpu, cv::cuda::PtrStepSz<uchar3> RGBMap_gpu, double paraAngle,
//	int cameraRows, int cameraCols, int depthThreshold, cv::cuda::PtrStepSz<uchar3>* ParaMat_gpu)
//{
//
//	//int coorX = blockIdx.x * blockDim.x + threadIdx.x;
//	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
//	int iCamera = blockIdx.z;
//
//	printf("thread %d successfully ran\n", coorY);
//
//	/*不满足阈值赋值为0*/
//	uchar3 zeros;
//	zeros.x = 0; zeros.y = 0; zeros.z = 0;
//	if (iCamera > blockDim.z / 2) {
//		/*按列遍历，每列同时异步处理*/
//		for (int iCoorX = 0; iCoorX < depthMap_gpu.cols; iCoorX++)
//		{
//			int coorParaX = (iCoorX - (int(depthMap_gpu.ptr(coorY)[iCoorX]) - 50) * paraAngle * (iCamera - cameraCols / 2));
//			int coorParaY = coorY;
//
//			if (iCoorX >= depthMap_gpu.cols || iCoorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
//			if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
//
//
//
//			/*如果小于深度阈值，则该像素直接舍弃*/
//			if (depthMap_gpu.ptr(coorY)[iCoorX] < depthThreshold) {
//				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = zeros;
//			}
//			else {
//				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[iCoorX];
//
//				/*空洞填补*/
//				if ((ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].x == zeros.x) &&
//					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].y == zeros.y) &&
//					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].z == zeros.z)) {
//					if (coorParaX >= 1) {
//						ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX - 1];
//					}
//				}
//			}
//		}
//	}
//	else {
//		/*按列遍历，每列同时异步处理*/
//		for (int iCoorX = depthMap_gpu.cols - 1; iCoorX >= 0; iCoorX--)
//		{
//			int coorParaX = (iCoorX - (int(depthMap_gpu.ptr(coorY)[iCoorX]) - 50) * paraAngle * (iCamera - cameraCols / 2));
//			int coorParaY = coorY;
//
//			if (iCoorX >= depthMap_gpu.cols || iCoorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
//			if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
//
//
//
//			/*如果小于深度阈值，则该像素直接舍弃*/
//			if (depthMap_gpu.ptr(coorY)[iCoorX] < depthThreshold) {
//				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = zeros;
//			}
//			else {
//				ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[iCoorX];
//
//				/*空洞填补*/
//				if ((ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].x == zeros.x) &&
//					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].y == zeros.y) &&
//					(ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX].z == zeros.z)) {
//					if (coorParaX >= 1) {
//						ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX] = ParaMat_gpu[iCamera].ptr(coorParaY)[coorParaX - 1];
//					}
//				}
//			}
//		}
//	}
//
//
//}


/*
* 函数名：GenParaImage_KERNEL
* 功能：CUDA并行计算生成单张视差图
* 输入：	
*	RGBMap_gpu	【原RGB图（GPU）】
*	depthMap_gpu	【经过预处理的Depth图（GPU）】
*	backgraound_gpu	【背景填充图（GPU）】
*	paraAngle	【相邻视点视角差】
*	cameraRows	【相机行数（二维集成成像EIA生成时用到）】
*	cameraCols	【相机列数】
*	iCamera		【视差图序号/相机索引】
*	depthThreshold	【深度阈值】
* 输出：
*	ParaMat_gpu	【视差图（GPU）】
*/
__global__ void GenParaImage_KERNEL(cv::cuda::PtrStepSz<uchar3> RGBMap_gpu, cv::cuda::PtrStepSz<uchar> depthMap_gpu, cv::cuda::PtrStepSz<uchar3> backgraound_gpu,
	double paraAngle, int cameraRows, int cameraCols, int iCamera, int depthThreshold, cv::cuda::PtrStepSz<uchar3> ParaMat_gpu)
{
	
	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
	///*测试用*/
	//printf("thread %d successfully ran\n", coorY);

	/*不满足阈值赋值为0*/
	uchar3 zeros;
	zeros.x = 0; zeros.y = 0; zeros.z = 0;
	/*按列遍历，每列同时异步处理*/
	int coorParaX = (coorX - (int(depthMap_gpu.ptr(coorY)[coorX]) + 50) * paraAngle * (iCamera - cameraCols / 2));
	int coorParaY = coorY;

	if (coorX >= depthMap_gpu.cols || coorX < 0 || coorY >= depthMap_gpu.rows || coorY < 0) return;
	if (coorParaX >= depthMap_gpu.cols || coorParaX < 0 || coorParaY >= depthMap_gpu.rows || coorParaY < 0) return;
		

	/*如果小于深度阈值，则该像素直接舍弃*/
	if (depthMap_gpu.ptr(coorY)[coorX] < depthThreshold) {
		ParaMat_gpu.ptr(coorParaY)[coorParaX] = 
			backgraound_gpu.ptr((int)(coorParaY*backgraound_gpu.rows/depthMap_gpu.rows))
			[(int)(coorParaX*backgraound_gpu.cols / depthMap_gpu.cols)];
	}
	else {
		ParaMat_gpu.ptr(coorParaY)[coorParaX] = RGBMap_gpu.ptr(coorY)[coorX];
	}		
}


/*
* 函数名：FillVacuum_forward_KERNEL
* 功能：视差图反向空洞填补
* 输入：ParaMat_gpu	【RGB视差图（GPU）】
* 输出：ParaMat_gpu 【RGB视差图（GPU）】
*/
__global__ void FillVacuum_backward_KERNEL(cv::cuda::PtrStepSz<uchar3> ParaMat_gpu) {
	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
	int coorY = blockIdx.y * blockDim.y + threadIdx.y;

	uchar3 zeros;
	zeros.x = 0; zeros.y = 0; zeros.z = 0;

	if ((ParaMat_gpu.ptr(coorY)[coorX].x == zeros.x)&&
		(ParaMat_gpu.ptr(coorY)[coorX].y == zeros.y)&&
		(ParaMat_gpu.ptr(coorY)[coorX].z == zeros.z)) {
		if (coorX < ParaMat_gpu.cols - 1) {
			ParaMat_gpu.ptr(coorY)[coorX] = ParaMat_gpu.ptr(coorY)[coorX + 1];
		}		
	}
}


/*
* 函数名：FillVacuum_forward_KERNEL
* 功能：视差图前向空洞填补
* 输入：ParaMat_gpu	【RGB视差图】
* 输出：ParaMat_gpu 【RGB视差图】
*/
__global__ void FillVacuum_forward_KERNEL(cv::cuda::PtrStepSz<uchar3> ParaMat_gpu) {
	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
	int coorY = blockIdx.y * blockDim.y + threadIdx.y;

	uchar3 zeros;
	zeros.x = 0; zeros.y = 0; zeros.z = 0;

	if ((ParaMat_gpu.ptr(coorY)[coorX].x == zeros.x) &&
		(ParaMat_gpu.ptr(coorY)[coorX].y == zeros.y) &&
		(ParaMat_gpu.ptr(coorY)[coorX].z == zeros.z)) {
		if (coorX > 0) {
			ParaMat_gpu.ptr(coorY)[coorX] = ParaMat_gpu.ptr(coorY)[coorX - 1];
		}
	}
}


/*
* 函数名：simplifiedFillMouth_KERNEL
* 功能：CUDA并行简易嘴部填充
* 输入：
*	depthImg	【原始深度图】
*	ctrPosX		【最大深度横坐标】
*	ctrPosY		【最大深度纵坐标】
* 输出：
*	depthImg	【直接在原图修改后输出】
*/
__global__ void simplifiedFillMouth_KERNEL(cv::cuda::PtrStepSz<uchar> depthGreySrc, int ctrPosX, int ctrPosY) {
	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
	if ((ctrPosY - coorY < 0) &&
		(sqrt(pow((double)(ctrPosY - coorY),2)+ pow((double)(ctrPosX - coorX), 2)) < 0.194 * depthGreySrc.rows) &&
		(abs((double)ctrPosX - coorX) < 0.0625 * depthGreySrc.cols)) {
		if (depthGreySrc.ptr(coorY)[coorX] <= 50) {
			depthGreySrc.ptr(coorY)[coorX] = 150;
		}
	}
}


/*
* 函数名：JointBilateralFilter_KERNEL
* 功能：CUDA并行双边滤波
* 输入：
*	colorImgSrc	【转为单通道的RGB图（GPU）】
*	depthImgSrc	【深度图（GPU）】
*	blackhatImg	【双边滤波限定区域蒙版（GPU）】
* 输出：
*	depthImgDst	【嘴部填充后的深度图（GPU）】
*/
__global__ void JointBilateralFilter_KERNEL(cv::cuda::PtrStepSz<uchar> colorImgSrc, cv::cuda::PtrStepSz<uchar> depthImgSrc, 
	cv::cuda::PtrStepSz<uchar> blackhatImg, cv::cuda::PtrStepSz<uchar> depthImgDst) {
	int coorX = blockIdx.x * blockDim.x + threadIdx.x;
	int coorY = blockIdx.y * blockDim.y + threadIdx.y;
	/*如果非0，那么是嘴部所处的连通区域，需要进行填补*/
	if (blackhatImg.ptr(coorY)[coorX] != 0) {
		///*对找出的嘴部进行等值化简化处理*/
		//depthImgDst.ptr(coorY)[coorX] = 150;
		//printf("一切正常，对(%d,%d)处像素进行处理\n", coorX, coorY);
		/*合适的距离和颜色权重*/
		double sigma_dis = 20;
		double sigma_color = 5;
		/*正方形邻域的大小*/
		int r = 30;
		/*获得[r,r]邻域的起点终点并防止越界*/
		int iMin = MAX(coorY - r, 0),
			iMax = MIN(coorY + r, depthImgSrc.rows - 1),
			jMin = MAX(coorX - r, 0),
			jMax = MIN(coorX + r, depthImgSrc.cols - 1);
		/*计算颜色和距离的权重*/
		double w_color, w_distance, w;
		double totalWeight = 0;
		double totalDepthValue = 0;
		for (int iNeighbour = iMin; iNeighbour <= iMax; iNeighbour++) {
			for (int jNeighbour = jMin; jNeighbour <= jMax; jNeighbour++) {
				w_color = exp(
					-pow((double)(colorImgSrc.ptr(iNeighbour)[jNeighbour] - colorImgSrc.ptr(coorY)[coorX])/255.0f, 2)/
					(double)(2 * pow(sigma_color, 2))
				);
				w_distance = exp(
					-(double)(pow((double)(iNeighbour - coorY), 2) + pow((double)(jNeighbour - coorX), 2)) /
					(double)(2 * pow(sigma_dis, 2))
				);
				w = w_color * w_distance;
				/*使邻域深度为0的像素点对目标像素无贡献*/
				totalWeight += w * (depthImgSrc.ptr(iNeighbour)[jNeighbour] == 0 ? 0 : 1);
				totalDepthValue += w * depthImgSrc.ptr(iNeighbour)[jNeighbour];
			}
		}
		depthImgDst.ptr(coorY)[coorX] = (int)(totalDepthValue / totalWeight);
		//printf("greyscale=\t%d\n", depthImgDst.ptr(coorY)[coorX]);
	}
}


/*
* 函数名：ParaImg2EIA_KERNEL
* 功能：将单张视差图按像素映射到EIA矩阵
* 输入：
*	inRGBMat_CUDA	【单张RGB视差图（GPU）】
*	iCurCam		【视差图序号/相机索引（GPU）】
*	effViewpointNum	【有效视点数】
*	slope		【光栅斜率】
* 输出：
*	outStereoMat_CUDA	【部分像素已经排列好的EIA（GPU）】
*/
__global__ void ParaImg2EIA_KERNEL(cv::cuda::PtrStepSz<uchar3> inRGBMat_CUDA, int iCurCam, double effViewpointNum, double slope, cv::cuda::PtrStepSz<uchar3> outStereoMat_CUDA)
{
	// 遍历子坐标
	int coorDim_X = blockIdx.x * blockDim.x + threadIdx.x;
	int coorDim_Y = blockIdx.y * blockDim.y + threadIdx.y;

	// 输入图像（视差图）坐标
	int coorIn_X = coorDim_X / 3 * inRGBMat_CUDA.cols / outStereoMat_CUDA.cols;
	int coorIn_Y = coorDim_Y * inRGBMat_CUDA.rows / outStereoMat_CUDA.rows;

	// 输出图像（合成图像）坐标
	int coorOut_X = coorDim_X / 3;
	int coorOut_Y = coorDim_Y;
	int isubx = coorDim_X % 3;

	float coordinate = 0.000000f;

	if (coorDim_Y % 2 == 0)	{
		coordinate = coorDim_X + coorDim_Y * 3 * slope + outStereoMat_CUDA.cols * 3;
	}
	else{
		coordinate = coorDim_X + coorDim_Y * 3 * slope + outStereoMat_CUDA.cols * 3 + (float)1 / 3;
	}

	int ifactor = effViewpointNum - 1 - floor(fmodf(coordinate, effViewpointNum));

	if (ifactor == iCurCam)	{
		if (coorIn_X >= 0 && coorIn_X < inRGBMat_CUDA.cols && coorIn_Y >= 0 && coorIn_Y < inRGBMat_CUDA.rows){
			if (coorOut_X >= 0 && coorOut_X < outStereoMat_CUDA.cols && coorOut_Y >= 0 && coorOut_Y < outStereoMat_CUDA.rows){
				if (isubx == 0)				{
					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].x = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].x;
				}
				else if (isubx == 1){
					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].y = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].y;
				}
				else{
					outStereoMat_CUDA.ptr(coorOut_Y)[coorOut_X].z = inRGBMat_CUDA.ptr(coorIn_Y)[coorIn_X].z;
				}
			}
		}
	}
}


/*
* 功能：简易嘴部填充
* 输入：
*	depthImg	【原始深度图】
*	ctrPosX		【最大深度横坐标】
*	ctrPosY		【最大深度纵坐标】
* 输出：
*	depthImg	【直接在原图修改后输出】
*/
extern "C" bool simplifiedFillMouth_CUDA(cv::cuda::GpuMat& depthImg, int ctrPosX, int ctrPosY) {
	dim3 blockSize(4, depthImg.rows / 5);
	dim3 gridSize(depthImg.cols / blockSize.x, 5);
	simplifiedFillMouth_KERNEL << <gridSize, blockSize >> > (depthImg, ctrPosX, ctrPosY);
	return true;
}


/*
* 功能：限定区域双边滤波
* 输入：
*	colorGreyImgSrc	【RGB图转化的灰度图】
*	depthImgSrc		【原始深度图】
*	blackhatImg		【底帽滤波结果，双边滤波限定区域蒙版】
* 输出：
*	depthImgDst		【输出滤波后深度图】
*/
extern "C" bool JointBilateralFilter_CUDA(cv::cuda::GpuMat& colorGreyImgSrc, cv::cuda::GpuMat& depthImgSrc, cv::cuda::GpuMat& blackhatImg, cv::cuda::GpuMat& depthImgDst) {
	dim3 blockSize(4, blackhatImg.rows / 5);
	dim3 gridSize(blackhatImg.cols / blockSize.x, 5);
	JointBilateralFilter_KERNEL << <gridSize, blockSize >> > (colorGreyImgSrc, depthImgSrc, blackhatImg, depthImgDst);
	return true;
}


/*
* 功能：生成单帧EIA
* 输入：
*	RGBMap_gpu	【原RGB图（GPU）】
*	depthMap_gpu	【经过预处理的Depth图（GPU）】
*	backgraound_gpu	【背景填充图（GPU）】
*	hardwarePara	【硬件参数】
*	paraImages_gpu	【视差图数组（GPU）】
* 输出：
*	eiaImage_gpu	【合成的EIA（GPU）】
*/
extern "C" bool genEIAImgs_CUDA(cv::cuda::GpuMat& RGBMap_gpu, cv::cuda::GpuMat& depthMap_gpu, cv::cuda::GpuMat& backgroundImg_gpu,
	HARDWAREPARA& hardwarePara, cv::cuda::GpuMat* &paraImages_gpu, cv::cuda::GpuMat& eiaImage_gpu)
{		
	/*五张视差图串行处理*/
	/*全并行处理未调通，但整个合成视差图和EIA时间不足1ms，不是主要消耗算力的过程*/
	for (int iCamera = 0; iCamera < hardwarePara._VIEWNUM; iCamera++)
	{
		/*合成视差图*/
		dim3 blockSize(4, depthMap_gpu.rows / 5);
		dim3 gridSize(depthMap_gpu.cols / blockSize.x, 5);

		GenParaImage_KERNEL << <gridSize, blockSize >> > (RGBMap_gpu, depthMap_gpu, backgroundImg_gpu, hardwarePara._DISPARITYANGLE, 1, hardwarePara._VIEWNUM,
			iCamera, hardwarePara._DEPTHTHRESHOLD, paraImages_gpu[iCamera]);
		
		/*空洞填补*/
		for (int iFillTimes = 0; iFillTimes < 2; iFillTimes++) {
			for (int iParaImg = 0; iParaImg < hardwarePara._VIEWNUM; iParaImg++) {
				if (iParaImg < hardwarePara._VIEWNUM / 2) {
					FillVacuum_backward_KERNEL << <gridSize, blockSize >> > (paraImages_gpu[iParaImg]);
				}
				else {
					FillVacuum_forward_KERNEL << <gridSize, blockSize >> > (paraImages_gpu[iParaImg]);
				}
			}
		}		
		
		/*将视差图RGB映射到EIA*/
		ParaImg2EIA_CUDA(paraImages_gpu[iCamera], iCamera, hardwarePara, eiaImage_gpu);
		//hipDeviceSynchronize();		
	}	
	return true;
}


/*
* 功能：将视差图像素映射到EIA上
* 输入：
*	inRGBMat_GPU	【RGB图像（GPU）】
*	iCurCam		【视差图的序号（GPU）】
*	hardwarePara	【硬件参数（GPU）】
* 输出：
*	outEiaMat_GPU	【EIA（GPU）】
*/
extern "C" bool ParaImg2EIA_CUDA(cv::cuda::GpuMat& inRGBMat_GPU, int iCurCam, HARDWAREPARA& hardwarePara, cv::cuda::GpuMat& outEiaMat_GPU)
{
	dim3 blockSize(30, 30);
	int pixel_W = outEiaMat_GPU.cols * outEiaMat_GPU.channels();;
	int pixel_H = outEiaMat_GPU.rows;
	dim3 gridSize(pixel_W/ blockSize.x, pixel_H /blockSize.y);

	ParaImg2EIA_KERNEL << <gridSize, blockSize >> > (inRGBMat_GPU, iCurCam, hardwarePara._EFFICIENTVIEWNUM
		, hardwarePara._EFFICIENTSLOPE, outEiaMat_GPU);
	return true;
}


